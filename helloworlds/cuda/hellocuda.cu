#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10 * 1000 * 1000

__global__ void vector_add(float *res, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++)
    {
        res[i] = a[i] + b[i];
    }
}

int main()
{
    float *a, *b, *res;
    float *d_a, *d_b, *d_res; 

    // Allocate memory
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    res = (float *)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_res, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(d_res, d_a, d_b, N);
    
    hipMemcpy(res, d_res, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        if(fabs(res[i] - a[i] - b[i]) > 0.00001) {
            printf("ERROR at %i: fabs(res[%i] (%f) - a[%i](%f) - b[%i](%f)) = %f\n", i, i, res[i], i, a[i], i, b[i], fabs(res[i] - a[i] - b[i]));
            exit(-1);
        }
    }
    printf("OK\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(res);
}
